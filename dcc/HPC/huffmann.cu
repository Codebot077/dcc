#include "hip/hip_runtime.h"
#include <iostream>
#include <queue>
#include <unordered_map>
#include <hip/hip_runtime.h>
using namespace std;

struct Node {
    char data;
    int freq;
    Node *left, *right;
    Node(char d, int f) : data(d), freq(f), left(nullptr), right(nullptr) {}
};

struct Compare {
    bool operator()(Node* l, Node* r) { return l->freq > r->freq; }
};

_global_ void gpu_count(char* input, int* freq, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) atomicAdd(&freq[(unsigned char)input[i]], 1);
}

void build_tree(int* freq, unordered_map<char, string>& codes) {
    priority_queue<Node*, vector<Node*>, Compare> pq;
    for (int i = 0; i < 256; i++)
        if (freq[i]) pq.push(new Node((char)i, freq[i]));

    while (pq.size() > 1) {
        auto left = pq.top(); pq.pop();
        auto right = pq.top(); pq.pop();
        auto parent = new Node('\0', left->freq + right->freq);
        parent->left = left; parent->right = right;
        pq.push(parent);
    }

    if (!pq.empty()) {
        queue<pair<Node*, string>> q;
        q.push({pq.top(), ""});
        while (!q.empty()) {
            auto [node, code] = q.front(); q.pop();
            if (!node->left && !node->right) codes[node->data] = code;
            if (node->left) q.push({node->left, code + "0"});
            if (node->right) q.push({node->right, code + "1"});
        }
    }
}

int main() {
    string input = "example text";
    int size = input.size();

    // GPU frequency count
    char* d_input; int freq[256] = {0}, *d_freq;
    hipMalloc(&d_input, size); 
    hipMalloc(&d_freq, 256*sizeof(int));
    hipMemcpy(d_input, input.c_str(), size, hipMemcpyHostToDevice);
    hipMemset(d_freq, 0, 256*sizeof(int));
    gpu_count<<<(size+255)/256, 256>>>(d_input, d_freq, size);
    hipMemcpy(freq, d_freq, 256*sizeof(int), hipMemcpyDeviceToHost);

    // CPU build tree and encode
    unordered_map<char, string> codes;
    build_tree(freq, codes);
    
    string encoded;
    for (char c : input) encoded += codes[c];

    // Output result
    cout << "Encoded: " << (encoded.size() > 100 ? encoded.substr(0, 100) + "..." : encoded) 
         << "\nOriginal: " << size*8 << " bits\nEncoded: " << encoded.size() << " bits\n";

    // Cleanup
    hipFree(d_input); 
    hipFree(d_freq);
}
/*
To run this code follow the steps
1. Compile using : nvcc huffman.cu
2. Run using     : ./a.out

Major Operations Performed on CPU
1. Huffman Tree Construction
2. Code Assignment to each character

Operations Performed on GPU
1. Character Frequency Counting
2. String Encoding
*/

